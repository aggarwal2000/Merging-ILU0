#include "hip/hip_runtime.h"
#include<iostream>
#include<stdio.h>
#include<vector>
#include<cassert>
#include<chrono>
#include<cmath>
#include "hip/hip_runtime_api.h"
#include "matrix.h"
#include "ReadWriteData.h"
#include "header.h"
#include "PrecondBiCGSTAB.h"
#include "SolverResults.h"
#include "factorization.h"



namespace {

#include "Bicgstab_common.hpp"


__device__ void legacy_sparse_lower_triangular_solve_mixed(const int num_rows, const int* const row_ptrs, const int* const col_inds, 
    const double* const factored_array_page, const int* const diag_info, const double* const vec_shared, volatile double* const temp_vec_shared)
{   
    
        const int row_index = threadIdx.x;

        if(row_index >= num_rows)
        {
            return;
        }

        double sum = 0;

        const int start = row_ptrs[row_index];
        const int end = diag_info[row_index];
        
        int i = start;
        
        
        bool completed = false;

        while(!completed)
        {   
            const int col_index = col_inds[i];

            if( i < end  &&  isfinite(temp_vec_shared[col_index]))
            {
                sum += factored_array_page[i] * temp_vec_shared[col_index];
                i++;
            }
            
            if(i == end)
            {   
                temp_vec_shared[row_index] = (vec_shared[row_index] - sum)/1; // 1 in place of factored_array_page[end]
                
                completed = true;
                
            }

            
        }

}


__device__ void legacy_sparse_upper_triangular_solve_mixed(const int num_rows, const int* const row_ptrs, const int* const col_inds, 
    const double* const factored_array_page, const int* const diag_info, volatile const double* const temp_vec_shared, volatile double* const vec_hat_shared)
{
    const int row_index = threadIdx.x;

    if(row_index >= num_rows)
    {
        return;
    }

    double sum = 0;

    const int start = diag_info[row_index];
    const int end = row_ptrs[row_index + 1]  - 1;
    int i = end;

    bool completed = false;

    while(!completed )
    {   
       

        const int col_index = col_inds[i];

        if( i > start && isfinite(vec_hat_shared[col_index]))
        {
            sum += factored_array_page[i] * vec_hat_shared[col_index];
            i--;
        }

      
        if(i == start)
        {
            vec_hat_shared[row_index] = (temp_vec_shared[row_index] - sum)/factored_array_page[start];
           
            completed = true;
        }

      
    }

}



__device__ void ApplyPreconditionerILU_mixed(const int num_rows, const int* const row_ptrs, const int* const col_inds, 
    const double* const factored_array_page, const int* const diag_info, const double* const vec_shared, volatile double* const vec_hat_shared)
{
     // vec_hat = precond * vec
    // => L * U  * vec_hat = vec
    // => L * y = vec , find y , and then U * vec_hat = y, find vec_hat

    //sparse triangular solves

    //if we want to use the busy waiting while loop approach, then the num_rows should be <= threadblock size, else there is possibility of a deadlock!
    assert(num_rows <= blockDim.x);
    //TODO: For upper trsv, use thread 0 for the bottommost row, this way we could avoid :  assert(num_rows <= blockDim.x), as there won't be a possibility of deadlock then!

    __shared__  volatile double temp_vec_shared[MAX_NUM_ROWS];

    for(int i = threadIdx.x ; i < num_rows; i += blockDim.x)
    {
        temp_vec_shared[i] = 1.8/0; //TODO: find a better way to deal with this!
        vec_hat_shared[i] = 1.3/0;

    }

    __syncthreads();
    
    
    legacy_sparse_lower_triangular_solve_mixed(num_rows, row_ptrs, col_inds, factored_array_page, diag_info, vec_shared, temp_vec_shared);

    __syncthreads();

    legacy_sparse_upper_triangular_solve_mixed(num_rows, row_ptrs, col_inds, factored_array_page, diag_info, temp_vec_shared, vec_hat_shared);
}

__device__ void fill_partial_current_row_array(const int nrows, const int curr_row_index, double* const current_row_elements_arr, const int* const row_ptrs, 
    const int* const col_idxs, const double* const page_values, const int* const diag_ptrs)
{
    const int diag_ele_loc = diag_ptrs[curr_row_index];
    const int row_end_loc = row_ptrs[curr_row_index + 1];


    for(int i = threadIdx.x + curr_row_index; i < nrows ; i += blockDim.x)
    {
        current_row_elements_arr[i] = 0;
    }

    __syncthreads();

    for(int loc = threadIdx.x + diag_ele_loc ; loc < row_end_loc ; loc += blockDim.x)
    {
        current_row_elements_arr[ col_idxs[loc] ] = page_values[ loc ];
    
    }

}

        
__device__ void modify_rows_below_curr_row(const int nrows, const int curr_row_index,const double* const column_elements_array_for_current_row, const int* const row_ptrs, 
    const int* const col_idxs, double* const page_values, const int* const diag_ptrs)
{       
    const int warp_id = threadIdx.x / WARP_SIZE;

    const int id_within_warp = threadIdx.x % WARP_SIZE;

    const int total_num_warps_in_block = blockDim.x / WARP_SIZE;

    __shared__ double row_ele_arr[MAX_NUM_ROWS];
    //initilaize it with zeroes

    for(int i = threadIdx.x + curr_row_index + 1; i < nrows ; i += blockDim.x)
    {
        row_ele_arr[i] = 0;
    }

    __syncthreads();
    
    //one warp per row
    for(int row_below_index = warp_id + curr_row_index + 1; row_below_index < nrows ; row_below_index += total_num_warps_in_block )
    {
        for(int i = id_within_warp + row_ptrs[row_below_index] ; i < row_ptrs[row_below_index + 1]; i += WARP_SIZE)
        {   
            const int col_index = col_idxs[i];
    
            if(col_index == curr_row_index)
            {   
                double diag_ele = page_values[diag_ptrs[curr_row_index]];
                assert(diag_ele != 0);
                double row_ele = page_values[i] / diag_ele;
                row_ele_arr[row_below_index] = row_ele;
                page_values[i] = row_ele;
            }
            
            __syncwarp(__activemask()); //else a warning

            if(col_index > curr_row_index)
            {
                double col_ele = column_elements_array_for_current_row[col_index];
                page_values[i] -= row_ele_arr[row_below_index] * col_ele; 

            }
            

        }

    }
    

}

    

__device__ void compute_exact_ilu_0_approach1(const int num_rows, const int num_nz, const int* const row_ptrs, 
    const int* const col_inds, double* const factored_array, const int* const diag_info)
{   
    
    const int page_id = blockIdx.x;

    __shared__ double current_row_elements_arr[MAX_NUM_ROWS];

    for(int curr_row_index = 0; curr_row_index < num_rows; curr_row_index++)
    {   
       
        fill_partial_current_row_array(num_rows, curr_row_index , current_row_elements_arr, row_ptrs, col_inds , factored_array +  num_nz * page_id, diag_info);

        __syncthreads();

        modify_rows_below_curr_row(num_rows, curr_row_index, current_row_elements_arr, row_ptrs, col_inds, factored_array + num_nz * page_id, diag_info);

        __syncthreads();

    }
}


__global__ void KernelBatchedPreconditionedBiCGSTAB_ILU(const int num_rows, const int num_nz, const int num_pages, const int* const row_ptrs, 
    const int* const col_inds, const double* const vals_mat, const double* const vals_rhs, double* const vals_ans,
    double* const factored_array, const int* const diag_info,float* const iter_counts , int* const conv_flags, 
    double* const iter_residual_norms)
{
  /* ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~shared memory ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~ */

   
    __shared__ double x_shared[MAX_NUM_ROWS];
    __shared__ double r_shared[MAX_NUM_ROWS];
    __shared__ double r_hat_shared[MAX_NUM_ROWS];
    __shared__ double p_shared[MAX_NUM_ROWS];
    __shared__ double v_shared[MAX_NUM_ROWS];
    __shared__ double s_shared[MAX_NUM_ROWS];
    __shared__ double t_shared[MAX_NUM_ROWS];
   // __shared__ double r_true_shared[MAX_NUM_ROWS];
    

    __shared__ double s_hat_shared[MAX_NUM_ROWS];
    __shared__ double p_hat_shared[MAX_NUM_ROWS];
 


    int page_id = blockIdx.x;


    if(page_id < num_pages)
    {   


        
        /*~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~ shared memory initialization/assigments~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~ */
        initialization(num_rows, num_nz, row_ptrs, col_inds, vals_mat, vals_rhs, x_shared, v_shared, p_shared, r_shared, r_hat_shared);
        
        __syncthreads();

        /*---------------------------------------------- generate preconditioner------------------------------------------*/

        for(int i = threadIdx.x; i < num_nz; i+= blockDim.x)
        {
            factored_array[i + page_id * num_nz] = vals_mat[i + page_id * num_nz];
        }

        __syncthreads();

        compute_exact_ilu_0_approach1(num_rows, num_nz, row_ptrs, col_inds, factored_array, diag_info);

        /*--------------------------------------------------- Preconditioner generated ----------------------------------------------------*/


        double res_initial = L2Norm(num_rows, r_shared); 
          
        double iter_residual_norm = res_initial;

        double rho_old = 1;
        double rho_new = 1;
        double omega_old = 1;
        double omega_new = 1;
        double alpha = 1;
        double beta = 1; 

        double b_norm = L2Norm(num_rows, vals_rhs + page_id*num_rows);
        
        int conv_flag = -1;


        if(b_norm == 0)
        {   
            for(int i = threadIdx.x; i < num_rows ; i += blockDim.x)
                x_shared[i] = 0;


            if(threadIdx.x == 0 )
            {   
                printf(" RHS for problem id: %d is 0. x = 0 is the solution. ",page_id);

                iter_counts[page_id] = 0;
                conv_flags[page_id] = 1;
                iter_residual_norms[page_id] = 0;
            }    

            __syncthreads();
        
        }
        else
        {
            if(res_initial < ATOL )
            {   
                if(threadIdx.x == 0 )
                {   
                    printf("\n Initial guess for problem id: %d is good enough. No need of iterations. \n", page_id);


                    iter_counts[page_id] = 0;
                    conv_flags[page_id] = 1;
                    iter_residual_norms[page_id] = res_initial;
                }	    
            }
            else
            {
                 /*~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~ Then can start iterating ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~ */
                    float iter = 0;
                
                    while(iter < MAX_ITER)
                    {
                        rho_new = inner_product(num_rows, r_shared, r_hat_shared);

                        if(rho_new == 0)
                        {
                            if(threadIdx.x == 0)
                            {
                                printf("\n Method failed for problem id: %d\n",page_id);
                            }

                            break;
                        }
                        
                        beta = (rho_new/rho_old)*(alpha/omega_old);
                    
                        
                        Update_p(num_rows,p_shared,r_shared ,v_shared,beta,omega_old);
                        __syncthreads();
                        

                        ApplyPreconditionerILU_mixed(num_rows, row_ptrs, col_inds, factored_array + page_id * num_nz, diag_info , p_shared, p_hat_shared);

                        __syncthreads();

                        SpMV(num_rows, row_ptrs,col_inds, vals_mat + page_id*num_nz, p_hat_shared, v_shared);
                        __syncthreads(); 
                        
                        
                        double r_hat_and_v_inner_prod = inner_product(num_rows,r_hat_shared,v_shared);
                        alpha = rho_new/r_hat_and_v_inner_prod;        
                    

                        Update_s(num_rows,s_shared,r_shared,alpha,v_shared);
                        __syncthreads();
                        

                        iter_residual_norm = L2Norm(num_rows, s_shared); //an estimate
                        
                        iter = iter + 0.5;

                      
                        if( iter_residual_norm < ATOL)
                        {
                            Update_x_middle(num_rows,x_shared,p_hat_shared,alpha);
                            __syncthreads();

                            conv_flag = 1;

                            
                            break;
    
                        }

                     
                        ApplyPreconditionerILU_mixed(num_rows, row_ptrs, col_inds, factored_array + page_id * num_nz, diag_info , s_shared, s_hat_shared);
                        __syncthreads();


                        SpMV( num_rows, row_ptrs , col_inds, vals_mat + page_id*num_nz , s_hat_shared, t_shared);
                        __syncthreads();
                    


                        double t_and_s_inner_prod = inner_product(num_rows,t_shared,s_shared);
                        double t_and_t_inner_prod = inner_product(num_rows,t_shared,t_shared);
                        omega_new = t_and_s_inner_prod/t_and_t_inner_prod;
                        

                        Update_x(num_rows,x_shared,p_hat_shared,s_hat_shared,alpha,omega_new);
                        __syncthreads();
                        
                        
                        iter = iter + 0.5;


                        Update_r(num_rows,r_shared,s_shared,t_shared,omega_new);
                        __syncthreads();

                        iter_residual_norm = L2Norm(num_rows,r_shared);
                        rho_old = rho_new;
                        omega_old = omega_new;

                        
                        if( iter_residual_norm < ATOL)
                        {   
                            conv_flag = 1;
                            break;
                        }

                        
                    }

                    __syncthreads();

                   /*  ComputeResidualVec(num_rows, row_ptrs , col_inds, vals_mat + page_id*num_nz, vals_rhs + page_id*num_rows, x_shared,r_true_shared);
                    __syncthreads();
                
                    
                    double true_resi_norm = L2Norm(num_rows,r_true_shared); */
                    
                    if(threadIdx.x == 0 )
                    {   
                      // printf("\nConv flag for problem_id: %d is %d , iter resi norm : %0.17lg, true resi norm: %0.17lg, iter:%f ",page_id,conv_flag, iter_residual_norm, true_resi_norm, iter );
                        iter_counts[page_id] = iter;
                        conv_flags[page_id] = conv_flag;
                        iter_residual_norms[page_id] = iter_residual_norm;
                    }

            }

        }

       
       // At the end,copy x_shared to global memory.
        for(int i = threadIdx.x; i < num_rows; i += blockDim.x)
            vals_ans[i + page_id*num_rows] = x_shared[i];

    
    }

}


int Batched_BiCGSTAB_ILU_Gpu_helper(const PagedCSRMatrices & A_pages,const PagedVectors& b_pages,PagedVectors & x_pages, 
    SolverResults & solver_results )
{
    std::cout << "\n\n-------------------------------------------------------------------------------\n Batched_Preconditioned BiCGSTAB_Gpu_helper " << std::endl;
    
   
    auto start = std::chrono::high_resolution_clock::now();
    
    //ilu preconditioner prep. phase- find diag ptrs 
    
    // (All these matrices are already sorted.(sorted while storing))
    int* diag_info = nullptr;
    hipMalloc((void**)&diag_info, sizeof(int) * A_pages.GetNumRows());

    int num_missing_diagonal_eles = Count_Missing_Diagonal_Elements(A_pages , diag_info);

    if(num_missing_diagonal_eles > 0)
    {
       assert(0);
    }

    Find_locations_of_diagonal_elements(A_pages, diag_info);
    
    double* factored_array = nullptr;
    hipMalloc((void**)&factored_array, sizeof(double)*A_pages.GetNumNz() * A_pages.GetNumPages());

    
    dim3 block(THREADS_PER_BLOCK,1,1);
    dim3 grid_solver(A_pages.GetNumPages(),1,1 );

    //------------------------------------------------------------------------------- Call main solver kernel-------------------------------------------------//


    KernelBatchedPreconditionedBiCGSTAB_ILU<<< grid_solver, block , 0  >>>(A_pages.GetNumRows(), A_pages.GetNumNz(), A_pages.GetNumPages(),
    A_pages.GetPtrToGpuRowPtrs(),A_pages.GetPtrToGpuColInd(), A_pages.GetPtrToGpuValues(), b_pages.GetPtrToGpuValues(), x_pages.GetPtrToGpuValues(),
    factored_array, diag_info , solver_results.GetPtrToGpuIterCount(), solver_results.GetPtrToGpuConvFlag() , solver_results.GetPtrToGpuIterResNorm());

    hipDeviceSynchronize();

    auto stop = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::microseconds>(stop - start);

    hipFree(diag_info);
    hipFree(factored_array);
    
    std::cout << " Time taken is: "  << (double)duration.count() << " microseconds\n\n ";  

    solver_results.SetTimeTaken((double)duration.count()/ 1000);


    //fill it with true residual norms
    KernelFillTrueResNorms<<< grid_solver , block , 0  >>>(A_pages.GetNumRows(), A_pages.GetNumNz(), A_pages.GetNumPages(), A_pages.GetPtrToGpuRowPtrs(),
    A_pages.GetPtrToGpuColInd(), A_pages.GetPtrToGpuValues(), b_pages.GetPtrToGpuValues(), x_pages.GetPtrToGpuValues(), solver_results.GetPtrToGpuTrueResNorm());

    hipDeviceSynchronize();

    return 1;
}



} //unnamed namespace


//----------------------------------------------------------------------------------------------------------------------------------------------------------------


// A*x = b
void Batched_conv_ILU_app1_Preconditioned_BiCGSTAB_merged_Gpu(const std::vector<std::string> & subdir, const PagedCSRMatrices & A_pages,
    const PagedVectors& b_pages,PagedVectors & x_pages,const bool is_scaled,  SolverResults & solver_results  )
{   

    assert(A_pages.ExistsGPU() == true);
    assert(b_pages.ExistsGPU() == true);
    assert(x_pages.ExistsGPU() == true);

    const int num_pages = A_pages.GetNumPages();
    assert(num_pages == b_pages.GetNumPages());
    assert(num_pages == x_pages.GetNumPages());

    const int num_rows = A_pages.GetNumRows();
    const int num_cols = A_pages.GetNumCols();
    
    assert(num_rows == num_cols);
    assert(num_cols == x_pages.GetNumElements());
    assert(num_rows == b_pages.GetNumElements());

   
    int success_code = 0;

    success_code = Batched_BiCGSTAB_ILU_Gpu_helper(A_pages,b_pages,x_pages, solver_results);

    std::string solution_file;

    if(is_scaled == true)
        solution_file = "x_scaled_gpu_conv_ilu_merged_app1_bicgstab.mtx";
    else
        solution_file = "x_gpu_conv_ilu_merged_app1_bicgstab.mtx";

    if(success_code == 1)
    {
        x_pages.CopyFromGpuToCpu();
        Print_ans(subdir,x_pages, solution_file);
        std::cout << "files containing soluation: x  are produced...  ( " <<  solution_file <<  " ) in their respective directories " << std::endl;

    }


}
